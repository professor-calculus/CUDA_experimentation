#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <ctime>
#include <algorithm>

// Thread block size
#define BLOCK_SIZE 1024

//  Size of Array
// #define SOA 67107840
// #define SOA 2147483647
#define SOA 1147483647
// #define SOA 8193

// Allocates an array with random integer entries.
void randomInit(int* data, unsigned long long int size)
{
	srand( time(0) );
	for (unsigned long long int i = 0; i < size; ++i)
	{
		data[i] = rand() & INT_MAX;
		//std::cout << data[i] << "\n";
	}
}

__global__ void ReductionMax2(int *input, int *results, unsigned long long int n)    //take thread divergence into account
{
	extern __shared__ int sdata[];
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tx = threadIdx.x;
	 //load input into __shared__ memory
	if(i < n)
	{
		sdata[tx] = input[i];
	}
	else
	{
		sdata[tx] = INT_MIN;
	}

	__syncthreads();

	// block-wide reduction
	for(unsigned int offset = 1; offset < blockDim.x; offset <<= 1)
	{
		int index = 2 * offset * tx;
		if(index < blockDim.x)
	        {
			// sdata[index] += sdata[index + offset];
			if(sdata[index] < sdata[index + offset]) sdata[index] = sdata[index + offset];
		}
		__syncthreads();
	}

	// finally, thread 0 writes the result
	if(threadIdx.x == 0)
	{
		// the result is per-block
		results[blockIdx.x] = sdata[0];
	}
}

// Reduce function wrapper
	void reduce(int* d_a, int* d_b)
	{
		unsigned long long int arraySize = SOA;
		unsigned long long int numBlocks = 1 + ((SOA - 1) / BLOCK_SIZE);

		int* device_intermediate;
		hipMalloc(&device_intermediate, sizeof(int)*numBlocks);
  		hipMemset(device_intermediate, 0, sizeof(int)*numBlocks);

		int i=1;

		do
		{
			std::cout << "GPU Iteration " << i << std::endl;
			i++;

			//setup execution parameters
                	dim3 block(BLOCK_SIZE);
	                dim3 grid(numBlocks);

        	        //execute the kernel
	                ReductionMax2<<<grid, block, BLOCK_SIZE*sizeof(int)>>>(d_a,device_intermediate,arraySize);
			arraySize = 1 + ((arraySize - 1) / BLOCK_SIZE);

			// device_in to device_intermediate
			hipMemcpy(d_a, device_intermediate, sizeof(int)*numBlocks, hipMemcpyDeviceToDevice);

			// Update required number of blocks
			numBlocks = 1 + ((numBlocks - 1) / BLOCK_SIZE);

			hipFree(device_intermediate);
			hipMalloc(&device_intermediate, sizeof(int)*numBlocks);
		}
		while(arraySize > BLOCK_SIZE);

		// Now compute the rest
		ReductionMax2<<<1, BLOCK_SIZE, BLOCK_SIZE*sizeof(int)>>>(d_a,d_b,arraySize);
	}


// get global max element via per-block reductions
	int main(int argc, char **argv)
	{
		// Introduce program
		std::cout << "Finding max element of an array on a GPU" << std::endl;

		// show memory usage of GPU
        	size_t free_byte ;
	        size_t total_byte ;
		hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

        	if ( hipSuccess != cuda_status )
		{
            		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
		        return 1;
		}

	        double free_db = (double)free_byte ;
        	double total_db = (double)total_byte ;
	        double used_db = total_db - free_db ;

        	std::cout << "GPU memory usage: used = " << used_db/1024.0/1024.0 << "MB, free = " <<
			free_db/1024.0/1024.0 << "MB, total = " << total_db/1024.0/1024.0 << " MB" << std::endl;



		// initial num of blocks
		unsigned long long int num_blocks = 1 + ((SOA - 1) / BLOCK_SIZE);
		std::cout << num_blocks << " blocks initially" << std::endl;

		//allocate host memory for array a
		unsigned long long int mem_size_a = sizeof(int) * SOA;
		if(mem_size_a > free_db)
		{
			std::cout << "Error: Not enough available GPU memory!" << std::endl;
			return 1;
		}

		std::cout << mem_size_a/1024.0/1024.0 << "MB requested" << std::endl;
		int* h_a = (int*)malloc(mem_size_a);

		//allocate device memory
		int* d_a;
		hipMalloc((void**) &d_a, mem_size_a);

		randomInit(h_a,SOA);

		//copy host memory to device
		hipMemcpy(d_a, h_a, mem_size_a, hipMemcpyHostToDevice);

		//allocate device memory for temporary results
		int mem_size_b = sizeof(int) * 1;
		int* d_b;
		hipMalloc((void**) &d_b, mem_size_b);
		int h_b;

		// Run our kernel wrapper
		reduce(d_a, d_b);

		//copy final result from device to host
		hipMemcpy(&h_b, d_b, sizeof(int), hipMemcpyDeviceToHost);

                std::cout << "GPU max: " << h_b << "\n";

		int max = *std::max_element(h_a, h_a + SOA);

		std::cout << "Old-fashioned way: " << max << "\n";

		//clean up memory
		free(h_a);
		hipFree(d_a);
		hipFree(d_b);

		hipDeviceReset();

}
